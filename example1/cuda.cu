#include "hip/hip_runtime.h"
#include "cuda.hpp"
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel for vector addition
// Each thread handles one element addition
__global__ void addKernel(const float* a, const float* b, float* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

__global__ void addKernel(const int* a, const int* b, int* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

// Host function to add two vectors on GPU using CUDA
void cudaVectorAdd(const std::vector<float>& a, const std::vector<float>& b,
                   std::vector<float>& result) {
    int n = a.size();
    if (n != b.size() || n != result.size()) {
        std::cerr << "Error: Vector sizes mismatch for CUDA float addition."
                  << std::endl;
        return;
    }

    float *dev_a, *dev_b, *dev_c;

    // Allocate GPU memory
    hipMalloc((void**)&dev_a, n * sizeof(float));
    hipMalloc((void**)&dev_b, n * sizeof(float));
    hipMalloc((void**)&dev_c, n * sizeof(float));

    // Copy data from host to GPU
    hipMemcpy(dev_a, a.data(), n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b.data(), n * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Launch CUDA kernel
    addKernel<<<numBlocks, blockSize>>>(dev_a, dev_b, dev_c, n);

    // Copy result from GPU to host
    hipMemcpy(result.data(), dev_c, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}

void cudaVectorAdd(const std::vector<int>& a, const std::vector<int>& b,
                   std::vector<int>& result) {
    int n = a.size();
    if (n != b.size() || n != result.size()) {
        std::cerr << "Error: Vector sizes mismatch for CUDA int addition."
                  << std::endl;
        return;
    }

    int *dev_a, *dev_b, *dev_c;

    // Allocate GPU memory
    hipMalloc((void**)&dev_a, n * sizeof(int));
    hipMalloc((void**)&dev_b, n * sizeof(int));
    hipMalloc((void**)&dev_c, n * sizeof(int));

    // Copy data from host to GPU
    hipMemcpy(dev_a, a.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b.data(), n * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Launch CUDA kernel
    addKernel<<<numBlocks, blockSize>>>(dev_a, dev_b, dev_c, n);

    // Copy result from GPU to host
    hipMemcpy(result.data(), dev_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
